#include "hip/hip_runtime.h"
#include "MapTest.cuh"

#include "..\String.cuh"

#include <math.h>

namespace MapTest
{
   __device__ void AssertEqual(int a, int b)
   {
      if (a != b) {
         printf("not equal: (%d, %d)\n", a, b);
      }
   }

   __device__ MapTest::MapTest()
   {
   }

   typedef Map::Map<int, int, Comparator::IntCompareFcn, Comparator::IntCompareFcn, Hasher::IntHashFcn, Hasher::IntHashFcn> IntTestType;
   typedef Map::Iterator<int, int, Comparator::IntCompareFcn, Comparator::IntCompareFcn, Hasher::IntHashFcn, Hasher::IntHashFcn> IntTestTypeItr;

   __device__ void MapTest::TestInteger()
   {
      int k = 0, v = 1;
      IntTestType m1 =
         CreateMapA<int, int, Comparator::IntCompareFcn, Comparator::IntCompareFcn, Hasher::IntHashFcn, Hasher::IntHashFcn>(k, v);
      int k0 = 1, v0 = 1;
      m1.Put(k0, v0);

      AssertEqual(m1.Size(), 2);

      IntTestType m2(m1);
      AssertEqual(m2.Size(), 2);
      int k1 = 2, v1 = 3;
      m2.Put(k1, v1);
      AssertEqual(m2.Size(), 3);
      AssertEqual(m1.Size(), 2);

      int c1 = 0;
      IntTestTypeItr itr1(m1);
      while (itr1.HasNext()) {
         //printf("(%d, %d) ", itr1.GetKey(), itr1.GetValue());
         c1++;
         itr1.Next();
      }
      AssertEqual(c1, 2);

      int c2 = 0;
      IntTestTypeItr itr2(m2);
      while (itr2.HasNext()) {
         //printf("(%d, %d) ", itr2.GetKey(), itr2.GetValue());
         c2++;
         itr2.Next();
      }
      AssertEqual(c2, 3);

      auto m3 = m2;
      AssertEqual(m3.Size(), 3);
      int k2 = 2, v2 = 3;
      m3.Put(k2, v2);
      AssertEqual(m3.Size(), 3);
      auto h2 = m2.HashCode();
      auto h3 = m3.HashCode();
      if (h2 != h3) {
         printf("HashCodes are different: %d, %d\n", h2, h3);
      }

      printf("MapTest::TestInteger() completed\n");
   }

   typedef Map::Map<String::String, double, String::CompareFcn, Comparator::DoubleCompareFcn, String::HashFcn, Hasher::DoubleHashFcn> StringTestType;
   typedef Map::Iterator<String::String, double, String::CompareFcn, Comparator::DoubleCompareFcn, String::HashFcn, Hasher::DoubleHashFcn> StringTestTypeItr;

   __device__ StringTestType makeMap()
   {
      StringTestType map;
      double v0 = 1, v1 = 2, v2 = 3, v3 = 4, v4 = 5;
      String::String
         A = String::String("A"),
         B = String::String("B"),
         C = String::String("C"),
         D = String::String("D"),
         E = String::String("E");
      map.Put(A, v0);
      map.Put(B, v1);
      map.Put(C, v2);
      map.Put(D, v3);
      map.Put(E, v4);
      return map;
   }

   __device__ void MapTest::TestString()
   {
      StringTestType m1;
      double v0 = 1, v1 = 2, v2 = 3, v3 = 4;
      String::String
         A = String::String("A"),
         B = String::String("B"),
         C = String::String("C"),
         D = String::String("D"),
         E = String::String("E");
      m1.Put(A, v0);
      m1.Put(B, v1);

      AssertEqual(m1.Size(), 2);

      StringTestType m2(m1);
      AssertEqual(m2.Size(), 2);
      m2.Put(C, v2);
      AssertEqual(m2.Size(), 3);
      AssertEqual(m1.Size(), 2);

      int c1 = 0;
      StringTestTypeItr itr1(m1);
      while (itr1.HasNext()) {
         //printf("(%s, %lf) ", itr1.GetKey().Get(), itr1.GetValue());
         c1++;
         itr1.Next();
      }
      AssertEqual(c1, 2);
      //printf("\n");

      int c2 = 0;
      StringTestTypeItr itr2(m2);
      while (itr2.HasNext()) {
         //printf("(%s, %lf) ", itr2.GetKey().Get(), itr2.GetValue());
         c2++;
         itr2.Next();
      }
      AssertEqual(c2, 3);
      //printf("\n");

      StringTestType m3 = m2;
      AssertEqual(m3.Size(), 3);
      m3.Put(D, v3);
      AssertEqual(m3.Size(), 4);

      int c3 = 0;
      StringTestTypeItr itr3(m3);
      while (itr3.HasNext()) {
         //printf("(%s, %lf) ", itr3.GetKey().Get(), itr3.GetValue());
         c3++;
         itr3.Next();
      }
      AssertEqual(m3.Size(), 4);
      //printf("\n");

      StringTestType map4;
      double v5 = ::sqrt(0.05), v6 = ::sqrt(0.04);
      String::String V1 = String::String("V1"), V2 = String::String("V2");
      map4.Put(V1, v5);
      map4.Put(V2, v6);

      int c4 = 0;
      StringTestTypeItr itr4(map4);
      while (itr4.HasNext()) {
         //printf("(%s, %lf) ", itr4.GetKey().Get(), itr4.GetValue());
         c4++;
         itr4.Next();
      }
      AssertEqual(map4.Size(), 2);
      AssertEqual(c4, 2);

      auto m5 = map4;
      if (!(m5 == map4)) {
         printf("maps are different\n");
      }
      if (!(m5.Size() == map4.Size())) {
         printf("maps sizes are different: %d, %d\n", map4.Size(), m5.Size());
      }
      if (m5.HashCode() != map4.HashCode()) {
         printf("maps hashcodes are different\n");
      }

      StringTestTypeItr itr5(m5);
      while (itr5.HasNext()) {
         //printf("(%s, %lf) ", itr5.GetKey().Get(), itr5.GetValue());
         itr5.Next();
      }

      auto m6 = makeMap();
      StringTestTypeItr itr6(m6);
      while (itr6.HasNext()) {
         //printf("(%s, %lf) ", itr6.GetKey().Get(), itr6.GetValue());
         itr6.Next();
      }
      //printf("\n");
      
      printf("MapTest::TestString() completed\n");
   }

   __device__ static bool doubleCmp(double& a, double& b) {
      return a == b;
   }

   class TestClass
   {
   public:
      __device__ TestClass() {}

      __device__ StringTestType& GetMap() { return m; }

      private:
      StringTestType m;
   };

   struct TestClassCompare
   {
      __device__ inline bool operator() (TestClass& lhs, TestClass& rhs)
      {
         return lhs.GetMap() == rhs.GetMap();
      }
   };

   struct TestClassHashFcn
   {
      __device__ inline unsigned int operator() (TestClass& t)
      {
         return t.GetMap().HashCode();
      }
   };

   //__device__  bool AreEqualTestClass(TestClass& a, TestClass& b)
   //{
   //   if (&a == &b) return true;
   //   //typedef bool(*pAreEqualStrings)(String::String&, String::String&);
   //   //pAreEqualStrings fcnptr = String::AreEqual;
   //   //printf("%p, %p\n", a.m.GetHasher(), Hasher::APHash);
   //   //printf("%p, %p\n", a.m.GetKeyCmp(), fcnptr);
   //   //printf("%p, %p\n", a.m.GetValCmp(), doubleCmp);

   //   //unsigned int tmp1 = 1;
   //   //auto h1 = Hasher::Hash((char*)&tmp1, sizeof(unsigned int));
   //   //unsigned int tmp2 = 1;
   //   //auto h2 = Hasher::APHash((char*)&tmp2, sizeof(unsigned int));
   //   //printf("value: (%d, %u), (%d, %u)\n", tmp1, h1, tmp2, h2);

   //   //auto vcmp = a.m.GetValCmp();
   //   //auto vcmp = doubleCmp;
   //   //double v1 = 1, v2 = 1;
   //   //if (vcmp(v1, v2)) {
   //   //   printf("AAAAAAAA\n");
   //   //}

   //   //printf("a size: %d\n", a.m.Size());
   //   //printf("b size: %d\n", b.m.Size());
   //   StringTestTypeItr itra(a.GetMap());
   //   //printf("A:\n");
   //   while (itra.HasNext()) {
   //      auto k = itra.GetKey();
   //      auto v = itra.GetValue();

   //      //printf("%s %lf\n", k.Get(), v);
   //      if (!b.GetMap().ContainsKey(k)) return false;
   //      //if (v != b.m.GetValue(k)) return false;
   //      itra.Next();
   //   }
   //   //printf("B:\n");
   //   //Map::Iterator<String::String, double> itrb(b.m);
   //   //while (itrb.HasNext()) {
   //   //   auto k = itrb.GetKey();
   //   //   auto v = itrb.GetValue();
   //   //   printf("%s %lf\n", k.Get(), v);
   //   //   itrb.Next();
   //   //}
   //   return true;
   //}

   typedef Map::Map<String::String, TestClass, String::CompareFcn, TestClassCompare, String::HashFcn, TestClassHashFcn> MapTestT;
   typedef Map::Iterator<String::String, TestClass, String::CompareFcn, TestClassCompare, String::HashFcn, TestClassHashFcn> MapTestTItr;

   __device__ void MapTest::TestMapOfMap()
   {
      MapTestT m1;
      TestClass a;
      TestClass b;

      double v1 = 1.2, v2 = 1.2, v3 = 3.3;

      a.GetMap().Put(String::String("A1"), v1);
      a.GetMap().Put(String::String("A2"), v2);
      a.GetMap().Put(String::String("A3"), v3);
      b.GetMap().Put(String::String("B1"), v1);
      m1.Put(String::String("A"), a);
      m1.Put(String::String("B"), b);

      printf("MapTest::TestMapOfMap() completed\n");
   }

   __device__ void MapTest::TestAggregate()
   {
      StringTestType t;
      double v0 = 0.1, v1 = 0.5, v2 = 100;
      t.Put(String::String("A"), v0);
      t.Put(String::String("B"), v1);
      t.Put(String::String("C"), v2);

      double ret1 = t.Aggregate([](double a) {return a*a; });
      double ret2 = t.Aggregate([](double a) {return a*a; });
      double ret3 = t.Aggregate([](double a) {return a*a; });
      double ans = 0.1*0.1 + 0.5*0.5 + 100 * 100;
      if (ret1 != ans) {
         printf("wrong : %lf, %lf", ret1, ans);
      }
      if (ret2 != ans) {
         printf("wrong : %lf, %lf", ret2, ans);
      }
      if (ret3 != ans) {
         printf("wrong : %lf, %lf", ret3, ans);
      }
      printf("MapTest::TestAggregate() completed\n");
   }
}